#include "hip/hip_runtime.h"
#include <iostream>
#include <stdlib.h>
#include "cuda_utils.h"

__global__ void render(float *fb, int maxX, int maxY) {
    int i = threadIdx.x + blockIdx.x + blockDim.x;
    int j = threadIdx.y + blockIdx.y + blockDim.y;
    if((i >= maxX) || (j >= maxY)) return;
    
    int pixelIdx = j*maxX*3 + i*3;
    fb[pixelIdx] = float(i) / maxX;
    fb[pixelIdx + 1] = float(j) / maxY;
    fb[pixelIdx + 2] = 0.2;
    
     
}


int main(int argc, char const *argv[]){
    if( argc < 3){
        std::cerr << "Missing arguments." << std::endl
                << "Usage: ./rayTracer <w> <h>" << std::endl;
        exit(-1);
    }
    
    int imageWidth = atoi(argv[1]);
    int imageHeight = atoi(argv[2]);
    int threadX = BLOCK_X;
    int threadY = BLOCK_Y;

    std:: cerr << "Rendering a " << imageWidth << "x" << imageHeight << " image in "
                << threadX << "x" << threadY << " blocks.\n";

    // Allocate frame buffer
    int numPixels = imageHeight * imageWidth;
    size_t frameBufferSize = 3* numPixels * sizeof(float);

    float *frameBuffer;

    checkCudaErrors(hipMallocManaged((void **)&frameBuffer, frameBufferSize));

    // Render
    dim3 blocks(imageWidth/threadX+1,imageHeight/threadY+1);
    dim3 threads(threadX,threadY);

    render<<<blocks,threads>>>(frameBuffer, imageWidth, imageHeight);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    std::cout << "P3\n" << imageWidth << " " << imageHeight <<"\n255\n";
    for(int j = imageHeight-1; j >= 0; j--){
        for(int i = 0; i < imageWidth; i++){
            size_t pixelIdx = 3*j*imageWidth + i*3;
            float r = frameBuffer[pixelIdx];
            float g = frameBuffer[pixelIdx + 1];
            float b = frameBuffer[pixelIdx + 2];

            int ir = int(255.99*r);
            int ig = int(255.99*g);
            int ib = int(255.99*b);
            std::cout << ir << " " << ig << " " << ib << "\n";
        }
    }
    checkCudaErrors(hipFree(frameBuffer));

    return 0;
}