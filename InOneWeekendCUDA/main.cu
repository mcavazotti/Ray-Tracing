#include "hip/hip_runtime.h"
#include <iostream>
#include <stdlib.h>
#include "cuda_utils.h"

__global__ void render(float *fb, int maxX, int maxY) {
    int i = threadIdx.x + blockIdx.x + blockDim.x;
    int j = threadIdx.y + blockIdx.y + blockDim.y;
    if((i >= maxX) || (j >= maxY)) return;
    
    int pixelIdx = j*maxX*3 + i*3;
    fb[pixelIdx] = float(i) / maxX;
    fb[pixelIdx + 1] = float(j) / maxY;
    fb[pixelIdx + 2] = 0.2;
    
     
}


int main(int argc, char const *argv[]){
    if( argc < 3){
        std::cerr << "Missing arguments." << std::endl
                << "Usage: ./rayTracer <h> <w>" << std::endl;
        exit(-1);
    }
    
    int imageWidth = atoi(argv[1]);
    int imageHeight = atoi(argv[2]);

    std:: cerr << "Rendering a " << imageWidth << "x" << imageHeight << " image in "
                << BLOCK_X << "x" << BLOCK_Y << " blocks.\n";

    // Allocate frame buffer
    int numPixels = imageHeight * imageWidth;
    size_t frameBufferSize = 3* numPixels * sizeof(float);

    float *frameBuffer;

    checkCudaErrors(hipMallocManaged(void **)&frameBuffer, frameBufferSize));

    // Render
    dim3 blocks(imageWidth/BLOCK_X+1,imageHeight/BLOCK_Y+1);
    dim3 threads(BLOCK_X,BLOCK_Y);

    render<<<blocks,threads>>>(frameBuffer, imageWidth, imageHeight);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    std::cout << "P3\n" << imageWidth << " " << imageHeight <<"\n255\n";
    for(int j = imageHeight-1; j >= 0; j--){
        for(int i = 0; i < imageWidth; i++){
            size_t pixelIdx = 3*j*imageWidth + i*3;
            float r = frameBuffer[pixelIdx];
            float g = frameBuffer[pixelIdx + 1];
            float b = frameBuffer[pixelIdx + 2];

            int ir = int(255.99*r);
            int ig = int(255.99*g);
            int ib = int(255.99*b);
            std::cout << ir << " " << ig << " " << ib << "\n";
        }
    }
    checkCudaErrors(hipFree(frameBuffer));

    return 0;
}