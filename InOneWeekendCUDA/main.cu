#include "hip/hip_runtime.h"

#include <iostream>
#include <hiprand/hiprand_kernel.h>
#include <float.h>
#include <time.h>

#include "cuda_utils.h"
#include "camera.h"
//#include "color.h"
#include "hittable_list.h"
#include "material.h"
#include "rtweekend.h"
#include "sphere.h"

__global__ void rand_init(hiprandState *randState) {
  if(threadIdx.x == 0 && blockIdx.x == 0)
    hiprand_init(1998, 0 ,0, randState);
}

__global__ void renderInit(int maxX, int maxY, hiprandState *randState) {
  int i = threadIdx.x + blockIdx.x*blockDim.x;
  int j = threadIdx.y + blockIdx.y*blockDim.y;

  if((i >= maxX) || (j >= maxY)) return;

  int pixelIdx = j*maxX + i;

  hiprand_init(1998 + pixelIdx,0,0, &randState[pixelIdx]);
}

__device__ color get_color(const ray &r, hittable **world, int maxRecursionDepth, hiprandState *localRandState){
  ray currentRay = r;
  color currentAttenuation = color(1,1,1);

  for(int i = 0; i < maxRecursionDepth; i++){
    hit_record rec;
    if((*world)->hit(currentRay, 0.001f, FLT_MAX, rec)){
      ray scattered;
      color attenuation;
      if(rec.mat_ptr->scatter(currentRay,rec,attenuation, scattered, localRandState)) {
        currentAttenuation = currentAttenuation * attenuation;
        currentRay = scattered;
      }
      else return color(0,0,0);
    }
    else {
      vec3 unitDirection = unit_vector(currentRay.direction());
      float t = 0.5f*(unitDirection.y() +1.0f);
      color c = (1.0f-t)*color(1.0, 1.0, 1.0) + t*color(0.5, 0.7, 1.0);

      return currentAttenuation * c;
    }
  }

  return color(0,0,0);
}

__global__ void render(color *fb, int maxX, int maxY, int samples,int recursionDepth, camera **cam, hittable **world, hiprandState *randState) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int j = threadIdx.y + blockIdx.y * blockDim.y;

  if((i >= maxX) || (j >= maxY)) return;

  int pixelIdx = j*maxX + i;

  hiprandState localRandState = randState[pixelIdx];
  color col(0,0,0);

  for(int s = 0; s < samples; s++){
    float u = float(i + hiprand_uniform(&localRandState)) / float(maxX);
    float v = float(j + hiprand_uniform(&localRandState)) / float(maxY);
    ray r = (*cam)->get_ray(u,v, &localRandState);
    col += get_color(r,world,recursionDepth,&localRandState);
  }

  randState[pixelIdx] = localRandState;

  col /= float(samples);
  col[0] = sqrtf(col[0]);
  col[1] = sqrtf(col[1]);
  col[2] = sqrtf(col[2]);

  fb[pixelIdx] = col;
}

__global__ void createWorld(hittable **d_list, hittable **d_world, camera **d_camera, int imgX, int imgY, hiprandState *currentState) {
  if(threadIdx.x == 0 && blockIdx.x == 0){
    hiprandState localRandState = *currentState;

    d_list[0] = new sphere(point3(0, -1000,-1), 1000, new lambertian(color(0.5,0.5,0.5)));
    int i = 1;
    
    for(int a = -11; a < 11; a++){
      for(int b = -11; b < 11; b++){
        float chooseMat = random_float(&localRandState);
        point3 center(a+random_float(&localRandState), 0.2, b+random_float(&localRandState));

        if(chooseMat < 0.8f)
          d_list[i++] = new sphere(center, 0.2, new lambertian(color(random_float(&localRandState)*random_float(&localRandState),random_float(&localRandState)*random_float(&localRandState),random_float(&localRandState)*random_float(&localRandState))));
        else if (chooseMat <0.95f)
          d_list[i++] = new sphere(center, 0.2, new metal(color(0.5f*(1.0f*random_float(&localRandState)),0.5f*(1.0f*random_float(&localRandState)),0.5f*(1.0f*random_float(&localRandState))), 0.5f*random_float(&localRandState)));
        else
          d_list[i++] = new sphere(center, 0.2, new dielectric(1.5));
      }
    }

    d_list[i++] = new sphere(point3(0,1,0), 1, new dielectric(1.5));
    d_list[i++] = new sphere(point3(-4,1,0), 1, new lambertian(color(0.4,0.2,0.1)));
    d_list[i++] = new sphere(point3(4,1,0), 1, new metal(color(0.7,0.6,0.5),0));

    *currentState = localRandState;

    *d_world = new hittable_list(d_list, 22*22+1+3);
    //*d_world = new hittable_list(d_list, 1+3);

    point3 lookFrom(13,2,3);
    point3 lookAt(0,0,0);

    float distToFocus = (lookFrom - lookAt).length();
    float aperture = 0.1;
    *d_camera = new camera(lookFrom, lookAt, vec3(0,1,0), 30, float(imgX)/float(imgY), aperture, distToFocus);
  }
}

__global__ void freeWorld(hittable **d_list, hittable **d_world, camera **d_camera) {
    for(int i=0; i < 22*22+1+3; i++) {
    //for(int i=0; i < 1+3; i++) {
        delete ((sphere *)d_list[i])->mat_ptr;
        delete d_list[i];
    }
    delete *d_world;
    delete *d_camera;
}

int main(int argc, char *argv[]) {
  if( argc < 4){
        std::cerr << "Missing arguments." << std::endl
                << "Usage: ./rayTracer <w> <h> <samples>" << std::endl;
        exit(-1);
    }
    
    int imageWidth = atoi(argv[1]);
    int imageHeight = atoi(argv[2]);
    int samplesPerPixel = atoi(argv[3]);

    float aspectRatio = imageWidth / imageHeight;
    int maxDepth = 50;
    int threadX = BLOCK_X;
    int threadY = BLOCK_Y;

    std:: cerr << "Rendering a " << imageWidth << "x" << imageHeight << " image in "
                << threadX << "x" << threadY << " blocks.\n";

    int numPixels = imageHeight * imageWidth;
    size_t frameBufferSize = numPixels * sizeof(color);

    color *frameBuffer;
    checkCudaErrors(hipMallocManaged((void **)&frameBuffer, frameBufferSize));

    hiprandState *d_randState;
    hiprandState *d_randState2;
    checkCudaErrors(hipMalloc((void **)&d_randState, numPixels*sizeof(hiprandState)));
    checkCudaErrors(hipMalloc((void **)&d_randState2, sizeof(hiprandState)));

    rand_init<<<1,1>>>(d_randState2);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    hittable **d_hittableList;
    int numHittables = 22*22+1+3;
    //int numHittables = 1+3;
    checkCudaErrors(hipMalloc((void **)&d_hittableList, numHittables * sizeof(hittable *)));
    
    hittable **d_world;
    checkCudaErrors(hipMalloc((void **)&d_world, sizeof(hittable *)));
    
    camera **d_camera;
    checkCudaErrors(hipMalloc((void **)&d_camera, sizeof(camera *)));

    createWorld<<<1,1>>>(d_hittableList, d_world, d_camera, imageWidth, imageHeight, d_randState2);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
  
    dim3 blocks(imageWidth/threadX +1, imageHeight/threadY + 1);
    dim3 threads(threadX,threadY);

    renderInit<<<blocks, threads>>>(imageWidth, imageHeight, d_randState);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    
    struct timespec start, stop;
    clock_gettime(CLOCK_MONOTONIC_RAW, &start);
    render<<<blocks, threads>>>(frameBuffer, imageWidth, imageHeight, samplesPerPixel, maxDepth, d_camera, d_world, d_randState);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    clock_gettime(CLOCK_MONOTONIC_RAW, &stop);
    double timer_milisecs =
      ((stop.tv_sec * 1000 * 1000 * 1000 + stop.tv_nsec) -
       (start.tv_sec * 1000 * 1000 * 1000 + start.tv_nsec))/(1000*1000) ;
    std::cerr << "Elapsed time " << timer_milisecs << "ms.\n";
    
    
    std::cout << "P3\n" << imageWidth << " " << imageHeight << "\n255\n";
    for (int j = imageHeight-1; j >= 0; j--) {
        for (int i = 0; i < imageWidth; i++) {
            size_t pixelIdx = j*imageWidth + i;
            int ir = int(255.99*frameBuffer[pixelIdx].x());
            int ig = int(255.99*frameBuffer[pixelIdx].y());
            int ib = int(255.99*frameBuffer[pixelIdx].z());
            std::cout << ir << " " << ig << " " << ib << "\n";
        }
    }

    checkCudaErrors(hipDeviceSynchronize());
    freeWorld<<<1,1>>>(d_hittableList,d_world,d_camera);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipFree(d_camera));
    checkCudaErrors(hipFree(d_world));
    checkCudaErrors(hipFree(d_hittableList));
    checkCudaErrors(hipFree(d_randState));
    checkCudaErrors(hipFree(d_randState2));
    checkCudaErrors(hipFree(frameBuffer));

    hipDeviceReset();

    std::cerr << "\nDone.\n";
    return 0;
  }
  